#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>


// BF16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16BF,
      lda,
      B,
      HIP_R_16BF,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

// FP32 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    float* A,
    int lda,
    float* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_32F,
      lda,
      B,
      HIP_R_32F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

template <typename T>
int wgrad_gemm_accum_fp32_cuda(T *input, T *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha = 1.0;
    const float beta  = 1.0;
    int status = 1;

    status = gemmex_wrapper(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        in_dim,
        out_dim,
        hidden_dim,
        &alpha,
        input,
        in_dim,
        d_output,
        out_dim,
        &beta,
        d_weight,
        in_dim);
    return status;
}

template int wgrad_gemm_accum_fp32_cuda<at::Half>(at::Half *input, at::Half *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template int wgrad_gemm_accum_fp32_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template int wgrad_gemm_accum_fp32_cuda<float>(float *input, float *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);
