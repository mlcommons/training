#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
//#include <hip/hip_runtime_api.h>
#include "THC/THC.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <math.h>
#include "/opt/pytorch/apex/apex/contrib/csrc/multihead_attn/softmax.h"

#define nstreams 16

// global variables.
hipStream_t stream[nstreams];
hipblasHandle_t handle;

///////////////////////////////////////////////////////////////////////////////////////////////////

void FastBmm1Fprop_(torch::Tensor &A,
                         torch::Tensor &B,
                         torch::Tensor &C,
	    	         int batch,
  	  	         torch::Tensor &seq_len,
                         int heads,
		         int embed,
			 bool scale,
			 bool strided,
			 bool enable_stream,
			 bool sync)
{

    float one = 1.0, zero = 0.0, alpha = 1.0 / sqrt(static_cast<float>(embed));

    int *seqlen = static_cast<int*>(seq_len.data_ptr());

    void *ptrA = static_cast<void*>(static_cast<half*>(A.data_ptr()) + (strided ? embed : 0)); 	// key
    void *ptrB = static_cast<void*>(static_cast<half*>(B.data_ptr())); 				// query
    void *ptrC = static_cast<void*>(static_cast<half*>(C.data_ptr())); 	        		// output

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        hipblasSetStream(handle, enable_stream ? stream[i%nstreams]: at::cuda::getCurrentCUDAStream());
        hipblasGemmStridedBatchedEx(handle,
                                   HIPBLAS_OP_T,
                                   HIPBLAS_OP_N,
                                   seqlen[i],
                                   seqlen[i],
                                   embed,
                                   static_cast<const void*>(scale ? &alpha : &one),
                                   ptrA,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   ptrB,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   static_cast<const void*>(&zero),
                                   ptrC,
                                   HIP_R_16F,
                                   seqlen[i],
                                   seqlen[i]*seqlen[i],
                                   enable_stream ? heads : batch*heads,
                                   HIP_R_32F,
                                   CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	ptrA = static_cast<void*>(static_cast<half*>(ptrA) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));
	ptrB = static_cast<void*>(static_cast<half*>(ptrB) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));
	ptrC = static_cast<void*>(static_cast<half*>(ptrC) + heads*seqlen[i]*seqlen[i]);
    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastBmm2Fprop_(torch::Tensor &A,
                    torch::Tensor &B,
                    torch::Tensor &C,
                    int batch,
                    torch::Tensor &seq_len,
                    int heads,
                    int embed,
		    bool scale,
		    bool strided,
		    bool enable_stream,
		    bool sync)
{

    float one = 1.0, zero = 0.0;

    int *seqlen = static_cast<int*>(seq_len.data_ptr());

    void *ptrA = static_cast<void*>(static_cast<half*>(A.data_ptr()) + (strided ? 2*embed : 0));  // value 
    void *ptrB = static_cast<void*>(static_cast<half*>(B.data_ptr()));            		// query*key
    void *ptrC = static_cast<void*>(static_cast<half*>(C.data_ptr()));           		 // output

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        hipblasSetStream(handle, enable_stream ? stream[i%nstreams]: at::cuda::getCurrentCUDAStream());
        hipblasGemmStridedBatchedEx(handle,
                                   HIPBLAS_OP_N,
                                   HIPBLAS_OP_N,
                                   embed,
                                   seqlen[i],
                                   seqlen[i],
                                   static_cast<const void*>(&one),
                                   ptrA,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   ptrB,
                                   HIP_R_16F,
                                   seqlen[i],
                                   seqlen[i]*seqlen[i],
                                   static_cast<const void*>(&zero),
                                   ptrC,
                                   HIP_R_16F,
                                   enable_stream ? heads*embed : batch*heads*embed,
                                   embed,
                                   enable_stream ? heads : batch*heads,
                                   HIP_R_32F,
                                   CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        ptrA = static_cast<void*>(static_cast<half*>(ptrA) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));
        ptrB = static_cast<void*>(static_cast<half*>(ptrB) + heads*seqlen[i]*seqlen[i]);
        ptrC = static_cast<void*>(static_cast<half*>(ptrC) + seqlen[i]*heads*embed);

    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastBmm1Dgrad1_(torch::Tensor &A,
                         torch::Tensor &B,
                         torch::Tensor &C,
                         int batch,
                         torch::Tensor &seq_len,
                         int heads,
                         int embed,
			 bool scale,
			 bool strided,
			 bool enable_stream,
			 bool sync)
{

    float one = 1.0, zero = 0.0, alpha = 1.0 / sqrt(static_cast<float>(embed));

    int *seqlen = static_cast<int*>(seq_len.data_ptr());

    void *ptrA = static_cast<void*>(static_cast<half*>(A.data_ptr()));           		// query
    void *ptrB = static_cast<void*>(static_cast<half*>(B.data_ptr()));
    void *ptrC = static_cast<void*>(static_cast<half*>(C.data_ptr()) + (strided ? embed : 0)); 	// grad_key

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        hipblasSetStream(handle, enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        hipblasGemmStridedBatchedEx(handle,
                                   HIPBLAS_OP_N,
                                   HIPBLAS_OP_T,
                                   embed,
                                   seqlen[i],
                                   seqlen[i],
                                   static_cast<const void*>(scale ? &alpha : &one),
                                   ptrA,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   ptrB,
                                   HIP_R_16F,
                                   seqlen[i],
                                   seqlen[i]*seqlen[i],
                                   static_cast<const void*>(&zero),
                                   ptrC,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   enable_stream ? heads : heads*batch,
                                   HIP_R_32F,
                                   CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        ptrA = static_cast<void*>(static_cast<half*>(ptrA) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));
        ptrB = static_cast<void*>(static_cast<half*>(ptrB) + heads*seqlen[i]*seqlen[i]);
        ptrC = static_cast<void*>(static_cast<half*>(ptrC) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));

    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastBmm2Dgrad1_(torch::Tensor &A,
                     torch::Tensor &B,
                     torch::Tensor &C,
                     int batch,
                     torch::Tensor &seq_len,
                     int heads,
                     int embed,
		     bool scale,
		     bool strided,
		     bool enable_stream,
		     bool sync)
{

    float one = 1.0, zero = 0.0;

    int *seqlen = static_cast<int*>(seq_len.data_ptr());

    void *ptrA = static_cast<void*>(static_cast<half*>(A.data_ptr()) + (strided ? 2*embed : 0));  // value
    void *ptrB = static_cast<void*>(static_cast<half*>(B.data_ptr()));
    void *ptrC = static_cast<void*>(static_cast<half*>(C.data_ptr()));

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        hipblasSetStream(handle, enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        hipblasGemmStridedBatchedEx(handle,
                                   HIPBLAS_OP_T,
                                   HIPBLAS_OP_N,
                                   seqlen[i],
                                   seqlen[i],
                                   embed,
                                   static_cast<const void*>(&one),
                                   ptrA,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   ptrB,
                                   HIP_R_16F,
				   enable_stream ? heads*embed : batch*heads*embed,
                                   embed,
                                   static_cast<const void*>(&zero),
                                   ptrC,
                                   HIP_R_16F,
                                   seqlen[i],
                                   seqlen[i]*seqlen[i],
                                   enable_stream ? heads : batch*heads,
                                   HIP_R_32F,
                                   CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        ptrA = static_cast<void*>(static_cast<half*>(ptrA) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));
        ptrB = static_cast<void*>(static_cast<half*>(ptrB) + seqlen[i]*heads*embed);
        ptrC = static_cast<void*>(static_cast<half*>(ptrC) + heads*seqlen[i]*seqlen[i]);

    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastBmm1Dgrad2_(torch::Tensor &A,
                         torch::Tensor &B,
                         torch::Tensor &C,
                         int batch,
                         torch::Tensor &seq_len,
                         int heads,
                         int embed,
			 bool scale,
			 bool strided,
			 bool enable_stream,
			 bool sync)
{

    float one = 1.0, zero = 0.0, alpha = 1.0 / sqrt(static_cast<float>(embed));

    int *seqlen = static_cast<int*>(seq_len.data_ptr());

    void *ptrA = static_cast<void*>(static_cast<half*>(A.data_ptr()) + (strided ? embed : 0));  	// key
    void *ptrB = static_cast<void*>(static_cast<half*>(B.data_ptr()));
    void *ptrC = static_cast<void*>(static_cast<half*>(C.data_ptr()));          		// grad query

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        hipblasSetStream(handle, enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        hipblasGemmStridedBatchedEx(handle,
                                   HIPBLAS_OP_N,
                                   HIPBLAS_OP_N,
                                   embed,
                                   seqlen[i],
                                   seqlen[i],
                                   static_cast<const void*>(scale ? &alpha : &one),
                                   ptrA,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   ptrB,
                                   HIP_R_16F,
                                   seqlen[i],
                                   seqlen[i]*seqlen[i],
                                   static_cast<const void*>(&zero),
                                   ptrC,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   enable_stream ? heads : batch*heads,
                                   HIP_R_32F,
                                   CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        ptrA = static_cast<void*>(static_cast<half*>(ptrA) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));
        ptrB = static_cast<void*>(static_cast<half*>(ptrB) + heads*seqlen[i]*seqlen[i]);
        ptrC = static_cast<void*>(static_cast<half*>(ptrC) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));

    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastBmm2Dgrad2_(torch::Tensor &A,
                     torch::Tensor &B,
                     torch::Tensor &C,
                     int batch,
                     torch::Tensor &seq_len,
                     int heads,
                     int embed,
		     bool scale,
		     bool strided,
		     bool enable_stream,
		     bool sync)
{

    float one = 1.0, zero = 0.0;

    int *seqlen = static_cast<int*>(seq_len.data_ptr());

    void *ptrA = static_cast<void*>(static_cast<half*>(A.data_ptr()));
    void *ptrB = static_cast<void*>(static_cast<half*>(B.data_ptr()));
    void *ptrC = static_cast<void*>(static_cast<half*>(C.data_ptr()) + (strided ? 2*embed : 0));  // grad-value

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        hipblasSetStream(handle, enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        hipblasGemmStridedBatchedEx(handle,
                                   HIPBLAS_OP_N,
                                   HIPBLAS_OP_T,
                                   embed,
                                   seqlen[i],
                                   seqlen[i],
                                   static_cast<const void*>(&one),
                                   ptrA,
                                   HIP_R_16F,
				   enable_stream ? heads*embed : batch*heads*embed,
                                   embed,
                                   ptrB,
                                   HIP_R_16F,
                                   seqlen[i],
                                   seqlen[i]*seqlen[i],
                                   static_cast<const void*>(&zero),
                                   ptrC,
                                   HIP_R_16F,
                                   (enable_stream ? 1 : batch) * (strided ? heads*3*embed : heads*embed),
                                   strided ? 3*embed : embed,
                                   enable_stream ? heads : batch*heads,
                                   HIP_R_32F,
                                   CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        ptrA = static_cast<void*>(static_cast<half*>(ptrA) + seqlen[i]*heads*embed);
        ptrB = static_cast<void*>(static_cast<half*>(ptrB) + heads*seqlen[i]*seqlen[i]);
        ptrC = static_cast<void*>(static_cast<half*>(ptrC) + (strided ? seqlen[i]*heads*3*embed : seqlen[i]*heads*embed));

    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastSoftmaxFprop_(torch::Tensor &input,
		  int batch,
                  torch::Tensor &seq_len,
		  int heads,
		  bool enable_stream,
		  bool sync)
{
    int *seqlen = static_cast<int*>(seq_len.data_ptr());
    void *ptrIn = static_cast<void*>(input.data_ptr());

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        dispatch_softmax<half, half, float>(
                                 reinterpret_cast<half*>(ptrIn),
                                 reinterpret_cast<const half*>(ptrIn),
                                 seqlen[i],
                                 seqlen[i],
                                 enable_stream ? heads*seqlen[i] : batch*heads*seqlen[i]);
        ptrIn = static_cast<void*>(static_cast<half*>(ptrIn) + heads*seqlen[i]*seqlen[i]);
    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastSoftmaxBprop_(torch::Tensor &input,
		       torch::Tensor &output,
                       int batch,
                       torch::Tensor &seq_len,
                       int heads,
		       bool enable_stream,
		       bool sync)
{
    int *seqlen = static_cast<int*>(seq_len.data_ptr());
    void *ptrIn = static_cast<void*>(input.data_ptr());
    void *ptrOut = static_cast<void*>(output.data_ptr());

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        dispatch_softmax_backward_stream<half, half, float>(
                                 static_cast<half*>(ptrOut),
                                 static_cast<half*>(ptrOut),
                                 reinterpret_cast<half const*>(ptrIn),
                                 seqlen[i],
                                 seqlen[i],
                                 enable_stream ? heads*seqlen[i] : batch*heads*seqlen[i], 
				 enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        ptrIn = static_cast<void*>(static_cast<half*>(ptrIn) + heads*seqlen[i]*seqlen[i]);
        ptrOut = static_cast<void*>(static_cast<half*>(ptrOut) + heads*seqlen[i]*seqlen[i]);	
    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }	
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastMaskSoftmaxFprop_(torch::Tensor &input,
                           torch::Tensor &mask,
                           int batch,
                           torch::Tensor &seq_len,
                           int heads,
			   bool enable_stream,
			   bool sync)
{
    int *seqlen = static_cast<int*>(seq_len.data_ptr());
    void *ptrIn = static_cast<void*>(input.data_ptr());
    void *ptrMask = static_cast<void*>(mask.data_ptr());

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        dispatch_additive_masked_softmax_stream<half, half, float>(
                                 reinterpret_cast<half*>(ptrIn),
                                 reinterpret_cast<const half*>(ptrIn),
                                 reinterpret_cast<const half*>(ptrMask),				 
                                 seqlen[i],
                                 seqlen[i],
                                 enable_stream ? heads*seqlen[i] : batch*heads*seqlen[i],
				 enable_stream ? heads*seqlen[i] : heads*seqlen[i], 
				 enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        ptrIn = static_cast<void*>(static_cast<half*>(ptrIn) + heads*seqlen[i]*seqlen[i]);
        ptrMask = static_cast<void*>(static_cast<half*>(ptrMask) + seqlen[i]);	
    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

std::vector<torch::Tensor> FastMaskSoftmaxDropoutFprop_(torch::Tensor &input,
                                  torch::Tensor &mask,
                                  int batch,
                                  torch::Tensor &seq_len,
                                  int heads,
                                  float dropout_prob,
                                  bool enable_stream,
                                  bool sync,
                                  bool is_training)
{
    int *seqlen = static_cast<int*>(seq_len.data_ptr());
    void *ptrIn = static_cast<void*>(input.data_ptr());
    void *ptrMask = static_cast<void*>(mask.data_ptr());

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        dispatch_additive_masked_softmax_stream<half, half, float>(
                                 reinterpret_cast<half*>(ptrIn),
                                 reinterpret_cast<const half*>(ptrIn),
                                 reinterpret_cast<const half*>(ptrMask),
                                 seqlen[i],
                                 seqlen[i],
                                 enable_stream ? heads*seqlen[i] : batch*heads*seqlen[i],
                                 enable_stream ? heads*seqlen[i] : heads*seqlen[i],
                                 enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        ptrIn = static_cast<void*>(static_cast<half*>(ptrIn) + heads*seqlen[i]*seqlen[i]);
        ptrMask = static_cast<void*>(static_cast<half*>(ptrMask) + seqlen[i]);
    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }

    int ntokens = seqlen[0];
    for(int i = 1; i < (enable_stream ? batch : 2); i++) {
        ntokens += seqlen[i];
    }
    auto act_options  = input.options().requires_grad(false);
    auto mask_options = act_options.dtype(torch::kUInt8);
    torch::Tensor dropout_results   = torch::empty({batch*heads, ntokens},   act_options);
    torch::Tensor dropout_mask      = torch::empty({batch*heads, ntokens},   mask_options);
    //torch::Tensor dropout_results   = torch::empty({batch*heads, seqlen[0], seqlen[0]},   act_options);
    //torch::Tensor dropout_mask      = torch::empty({batch*heads, seqlen[0], seqlen[0]},   mask_options);
    if (is_training) {
        //use at:: function so that C++ version generates the same random mask as python version
        auto dropout_tuple = at::_fused_dropout(input, 1.0f-dropout_prob);
        dropout_results = std::get<0>(dropout_tuple);
        dropout_mask = std::get<1>(dropout_tuple);
    }
    return {dropout_results, dropout_mask};
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void FastMaskSoftmaxDropoutBprop_(torch::Tensor &input,
                              torch::Tensor &output,
                              torch::Tensor &dropout_mask,
                              int batch,
                              torch::Tensor &seq_len,
                              int heads,
                              float dropout_prob,
                              bool enable_stream,
                              bool sync)
{
    int *seqlen = static_cast<int*>(seq_len.data_ptr());
    void *ptrIn = static_cast<void*>(input.data_ptr());
    void *ptrOut = static_cast<void*>(output.data_ptr());
    void *ptrDropoutMask = static_cast<void*>(dropout_mask.data_ptr());

    for(int i = 0; i < (enable_stream ? batch : 1); i++) {
        dispatch_masked_scale_softmax_backward_stream<half, half, float, false>(
                                 static_cast<half*>(ptrOut),
                                 static_cast<half*>(ptrOut),
                                 reinterpret_cast<half const*>(ptrIn),
                                 reinterpret_cast<uint8_t const*>(ptrDropoutMask),
                                 1.0/(1.0-dropout_prob),
                                 seqlen[i],
                                 seqlen[i],
                                 enable_stream ? heads*seqlen[i] : batch*heads*seqlen[i],
                                 enable_stream ? stream[i%nstreams] : at::cuda::getCurrentCUDAStream());
        ptrIn = static_cast<void*>(static_cast<half*>(ptrIn) + heads*seqlen[i]*seqlen[i]);
        ptrOut = static_cast<void*>(static_cast<half*>(ptrOut) + heads*seqlen[i]*seqlen[i]);
    }
    for(int i = 0; i < (enable_stream ? nstreams : 0); i++) {
        if(sync) hipStreamSynchronize(stream[i]);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

void init_mha_cuda_extension()
{
    // CUDA Stream.
    for(int i = 0; i < nstreams; i++) {
        hipStreamCreate(&stream[i]);
    }

    // CuBlas Handle.
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("InitMHACUDAExtension", &init_mha_cuda_extension, "InitMHACUDAExtension");
  m.def("FastBmm1Fprop", &FastBmm1Fprop_, "FastBmm1Fprop");
  m.def("FastBmm1Dgrad1", &FastBmm1Dgrad1_, "FastBmm1Dgrad1"); 
  m.def("FastBmm1Dgrad2", &FastBmm1Dgrad2_, "FastBmm1Dgrad2"); 
  m.def("FastBmm2Fprop", &FastBmm2Fprop_, "FastBmm2Fprop");
  m.def("FastBmm2Dgrad1", &FastBmm2Dgrad1_, "FastBmm2Dgrad1");
  m.def("FastBmm2Dgrad2", &FastBmm2Dgrad2_, "FastBmm2Dgrad2");
  m.def("FastSoftmaxFprop", &FastSoftmaxFprop_, "FastSoftmaxFprop");
  m.def("FastSoftmaxBprop", &FastSoftmaxBprop_, "FastSoftmaxBprop");
  m.def("FastMaskSoftmaxFprop", &FastMaskSoftmaxFprop_, "FastMaskSoftmaxFprop");
  m.def("FastMaskSoftmaxDropoutFprop", &FastMaskSoftmaxDropoutFprop_, "FastMaskSoftmaxDropoutFprop");  
  m.def("FastMaskSoftmaxDropoutBprop", &FastMaskSoftmaxDropoutBprop_, "FastMaskSoftmaxDropoutBprop");
}
